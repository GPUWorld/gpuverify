
#include <hip/hip_runtime.h>
//xfail:BUGLE_ERROR
//--gridDim=1 --blockDim=32

//This kernel is racy: memset is called with variable length.
#define memset(dst,val,len) __builtin_memset(dst,val,len)

__global__ void kernel(uint4 *out) {
  uint4 vector; int len;
  memset(&vector, 0, len);
  out[threadIdx.x] = vector;
}